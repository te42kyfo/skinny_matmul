#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <algorithm>
#include <complex>
#include <cstdlib>
#include <functional>
#include <iomanip>
#include <iostream>
#include <vector>

#include "../cu_complex.h"
#include "../gpu_error.cuh"
#include "cublas.cuh"
#include "fix1.cuh"
#include "fix2.cuh"
#include "fix_blend.cuh"
#include "fix_fb.cuh"
#include "var1.cuh"

#if !defined PARM || !defined PARN
#error "PARM or PARN is not specified! Specify M and N to measure"
#endif

using namespace std;

#define XSTR(s) STR(s)
#define STR(s) #s

#ifdef FC
typedef complex<float> htype;
typedef hipFloatComplex dtype;
dtype makeDtype(htype v) { return make_hipFloatComplex(v.real(), v.imag()); }
#define RAND_HTYPE(gen) htype(gen, gen)
#define MAKE_DTYPE(v1, v2) make_hipFloatComplex(v1, v2)
string mode = "float complex";
int flopsPerCell = 8;

#elif DC
typedef complex<double> htype;
typedef hipDoubleComplex dtype;
dtype makeDtype(htype v) { return make_hipDoubleComplex(v.real(), v.imag()); }
#define RAND_HTYPE(gen) htype(gen, gen)
#define MAKE_DTYPE(v1, v2) make_hipDoubleComplex(v1, v2)
string mode = "double complex";
int flopsPerCell = 8;

#elif FR
typedef float htype;
typedef float dtype;
dtype makeDtype(htype v) { return v; }
#define RAND_HTYPE(gen) htype(gen)
#define MAKE_DTYPE(v1, v2) float(v1)
string mode = "float real";
int flopsPerCell = 2;

#elif DR
typedef double htype;
typedef double dtype;
dtype makeDtype(htype v) { return v; }
#define RAND_HTYPE(gen) htype(gen)
#define MAKE_DTYPE(v1, v2) double(v1)
string mode = "double real";
int flopsPerCell = 2;

#endif

using MatmulFunctionType = function<bool(
    const size_t, const int, const int, const int, const dtype*, const int,
    const dtype, const dtype*, const int, const dtype, dtype*, const int)>;

double dtime() {
  double tseconds = 0;
  struct timeval t;
  gettimeofday(&t, NULL);
  tseconds = (double)t.tv_sec + (double)t.tv_usec * 1.0e-6;
  return tseconds;
}

__global__ void initKernel(dtype* A, size_t N) {
  size_t tidx = blockDim.x * blockIdx.x + threadIdx.x;

  for (size_t idx = tidx; idx < N; idx += blockDim.x * gridDim.x) {
    A[idx] = MAKE_DTYPE(idx % 3 - 1, 0);
  }
}

dtype* A;
dtype* B;
dtype* C;

void initMatmul(int M, int N, int K, int lda, int ldb, int ldc,
                size_t blockCount) {
  GPU_ERROR(hipMalloc(&A, sizeof(dtype) * lda * K));
  GPU_ERROR(hipMalloc(&B, sizeof(dtype) * ldb * M));
  GPU_ERROR(hipMalloc(&C, sizeof(dtype) * ldc * K));
  initKernel<<<52, 256>>>(A, lda * K);
  initKernel<<<52, 256>>>(B, ldb * M);
  initKernel<<<52, 256>>>(C, ldc * K);
}

void deInitMatmul() {
  GPU_ERROR(hipFree(A));
  GPU_ERROR(hipFree(B));
  GPU_ERROR(hipFree(C));
}

double measureMatmul(MatmulFunctionType matmulFunction, size_t M, size_t N,
                     size_t K, int lda, int ldb, int ldc, size_t blockCount,
                     int iters, bool self, dtype beta) {
  GPU_ERROR(hipDeviceSynchronize());

  bool passed = true;
  double t1 = dtime();
  for (int iter = 0; iter < iters; iter++) {
    if (self)
      passed = matmulFunction(blockCount, M, N, K, C, ldc, makeDtype(2.0), B,
                              ldb, makeDtype(beta), C, ldc);
    else
      passed = matmulFunction(blockCount, M, N, K, A, lda, makeDtype(2.0), B,
                              ldb, makeDtype(beta), C, ldc);
  }
  GPU_ERROR(hipDeviceSynchronize());
  double t2 = dtime();
  double time = (t2 - t1) / iters;

  if (!passed)
    return -time;
  else
    return time;
}

int main(int argc, char** argv) {
  if (PARM == 0 || PARN == 0) {
    std::cout << "  M   N  name       K  blockcount     time  GFlop  GByte\n";
    return 0;
  }
  int m1 = 0;
  int m2 = 0;
  int n1 = 0;
  int n2 = 0;
  if (argc == 2) {
    m1 = 1;
    m2 = stoi(argv[1]);
  }
  if (argc >= 3) {
    m1 = stoi(argv[1]);
    m2 = stoi(argv[2]);
  }
  if (argc == 4) {
    cout << "Incomplete set of arguments\n";
    exit(1);
  }
  if (argc == 5) {
    n1 = stoi(argv[3]);
    n2 = stoi(argv[4]);
  }
  if (argc == 1) {
    m1 = m2 = PARM;
    n1 = n2 = PARN;
  }

  vector<pair<MatmulFunctionType, string>> versions;

#if PARM != 0 && PARN != 0
#ifdef FIX_BLEND
  versions.push_back({tsmm_fix_blend<dtype, PARM, PARN>, "FBLEND"});
#endif
#ifdef FIX_FB
  versions.push_back({tsmm_fix_fb<dtype, PARM, PARN>, "FIX_FB"});
#endif
#ifdef FIX1
  versions.push_back({tsmm_fix1<dtype, PARM, PARN>, "FIX_V1"});
#endif
#ifdef FIX2
  versions.push_back({tsmm_fix2<dtype, PARM, PARN>, "FIX_V2"});
#endif
#ifdef CUBLAS
  versions.push_back({tsmm_cublas<dtype>, "CUBLAS"});
#endif
#ifdef VAR1
  versions.push_back({tsmm_var1<dtype>, "VAR_V1"});
#endif
#endif

  initMatmul(1, 1, 2 * ((size_t)1 << 30) / ((1 + 1) * 8), 1, 1, 1, 8 * 13);

  for (int M = m1; M <= m2; M++) {
    for (int N = n1; N <= n2; N++) {
      if (n1 == 0 && n2 == 0) N = M;

      size_t maxK = 2 * ((size_t)1 << 30) / ((M + N) * 8);
      size_t K = 200;

      // One warmup call
      measureMatmul(versions[0].first, M, N, K, M, N, N, 13, 1, true, -1.0);
      double resultTime =
          measureMatmul(versions[0].first, M, N, K, M, N, N, 13, 1, true, -1.0);

      while (resultTime < 0.005 && K < maxK) {
        K = min(maxK, 2 * K);
        resultTime = measureMatmul(versions[0].first, M, N, K, M, N, N, 13, 1,
                                   true, -1.0);
      }

      for (const auto& matmulVersion : versions) {
        for (int self = 0; self <= (M == N) ? 1 : 0; self++) {
          for (htype beta = 0.0; beta <= 1.0; beta += 1.0) {
            int iters = 1;

            size_t lda = M;
            double bestTime = -1;
            int bestBlockCount = 0;
            for (int blockCount = 1 * 13; blockCount <= 8 * 13;
                 blockCount += 13) {
              int sampleSize = 3;
              vector<double> times(sampleSize);
              for (int t = 0; t < sampleSize; t++) {
                times[t] =
                    measureMatmul(matmulVersion.first, M, N, K, lda, N, N,
                                  blockCount, iters, (self == 1), beta);
              }
              times.erase(remove_if(begin(times), end(times),
                                    [](double time) { return time < 0; }),
                          end(times));
              sort(times.begin(), times.end());

              if (times.size() != 0 &&
                  (times[sampleSize / 2] < bestTime || bestBlockCount == 0)) {
                bestTime = times[sampleSize / 2];
                bestBlockCount = blockCount;
              }
            }
            double flops = 0;
            double bw = 0;

            if (bestTime > 0) {
              flops = (M + (beta == 0 ? 0 : 1)) * K * N * flopsPerCell /
                      bestTime * 1.0e-9;
              bw = ((beta == 0 || self == 1 ? 1.0 : 2.0) * N + M) * K *
                   sizeof(double) / bestTime * 1.0e-9;
            }
            cout << setw(3) << M << " " << setw(3) << N << " " << beta << " "
                 << (self == 1 ? "A*A" : "A*B") << " " << matmulVersion.second
                 << " " << setw(9) << K << "  " << setw(10) << bestBlockCount
                 << " " << setprecision(3) << setw(8) << bestTime << " "
                 << setw(5) << setprecision(3) << flops << " " << setw(5) << bw
                 << "\n";
            cout.flush();
          }
        }
      }
      if (versions.size() > 1) cout << "\n";
    }
  }
  deInitMatmul();
}
