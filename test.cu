#include <iostream>
#include <cstdlib>
#include <sys/time.h>
#include <iomanip>
#include <algorithm>
#include <vector>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <random>
#include <complex>

#include "skyblas.cuh"

#if !defined PARM || !defined PARN
#error "PARM or PARN is not specified! Specify M and N to test for"
#endif

using namespace std;

typedef double realt;
typedef complex<realt> htype;
typedef hipDoubleComplex dtype;

double dtime() {
  double tseconds = 0;
  struct timeval t;
  gettimeofday(&t, NULL);
  tseconds = (double)t.tv_sec + (double)t.tv_usec * 1.0e-6;
  return tseconds;
}

#define GPU_ERROR(ans) \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    cerr << "GPUassert: \"" << hipGetErrorString(code) << "\"  in " << file
         << ": " << line << "\n";
    if (abort) exit(code);
  }
}

void cpuDgemm(const Skyblas::MEMORY_ORDER AOrder,
              const Skyblas::MEMORY_ORDER BOrder, const size_t M,
              const size_t N, const size_t K, const htype alpha, const htype *A,
              const int lda, const htype *B, const int ldb, const htype beta,
              htype *C, const int ldc) {
#pragma omp parallel for
  for (size_t m = 0; m < M; m++) {
    for (size_t n = 0; n < N; n++) {
      htype sum = 0;
      for (size_t k = 0; k < K; k++) {
        sum += A[k * lda + m] * B[k * ldb + n];
      }
      C[n * ldc + m] = C[n * ldc + m] * beta + alpha * sum;
    }
  }
}

void printMatrix(vector<htype> m1, vector<htype> m2, size_t N, size_t M,
                 size_t ldc, string matchColor = "\e[32m",
                 string mismatchColor = "\e[31m") {
  for (size_t n = 0; n < N; n++) {
    for (size_t m = 0; m < M; m++) {
      if (m1[n * ldc + m] == m2[n * ldc + m])
        cout << matchColor;
      else
        cout << mismatchColor;

      cout << m1[n * ldc + m] << "\e[0m\t";
    }
    cout << "\n";
  }
}

template <typename cT, typename sT>
cT makeDtype(sT hval) {
  return hval;
}

template <>
hipDoubleComplex makeDtype<hipDoubleComplex, double>(double hval) {
  return make_hipDoubleComplex(hval, 0);
}
template <>
hipFloatComplex makeDtype<hipFloatComplex, float>(float hval) {
  return make_hipFloatComplex(hval, 0);
}

template <typename cT, typename sT>
cT randHtype(sT v1, sT v2) {
  return v1;
}

template <>
complex<float> randHtype<complex<float>, float>(float v1, float v2) {
  return complex<float>(v1, v2);
}

template <>
complex<double> randHtype<complex<double>, double>(double v1, double v2) {
  return complex<double>(v1, v2);
}

bool testMatmul(Skyblas::MEMORY_ORDER AOrder, Skyblas::MEMORY_ORDER BOrder,
                size_t M, size_t N, size_t K, int lda, int ldb, int ldc,
                size_t blockCount, bool self) {
  dtype *A, *B, *d_temp_storage, *C;

  htype halpha = 1.0;
  htype hbeta = 2.0;

  dtype dalpha = makeDtype<dtype, realt>(1.0);
  dtype dbeta = makeDtype<dtype, realt>(2.0);

  cout.flush();
  GPU_ERROR(hipMalloc(&A, sizeof(dtype) * lda * K));
  GPU_ERROR(hipMalloc(&B, sizeof(dtype) * ldb * K));
  GPU_ERROR(hipMalloc(&C, sizeof(dtype) * ldc * N));

  vector<htype> hA(lda * K);
  vector<htype> hB(ldb * K);
  vector<htype> hB2(ldb * K);
  vector<htype> hC(ldc * N, 0);
  vector<htype> hC2(ldc * N, 0);
  vector<htype> cpuC(ldc * N, 0);

#pragma omp parallel
  {
    random_device r;
    default_random_engine gen(r());
    uniform_int_distribution<int> dis(-2, 2);
#pragma omp for
    for (size_t i = 0; i < lda * K; i++) {
      hA[i] = randHtype<htype, realt>(dis(gen), dis(gen));
    }
#pragma omp for
    for (size_t i = 0; i < ldb * K; i++) {
      hB[i] = randHtype<htype, realt>(dis(gen), dis(gen));
    }
#pragma omp for
    for (size_t i = 0; i < ldc * N; i++) {
      hC2[i] = hC[i] = cpuC[i] = randHtype<htype, realt>(dis(gen), dis(gen));
    }
  }
  GPU_ERROR(
      hipMemcpy(A, hA.data(), sizeof(htype) * lda * K, hipMemcpyDefault));
  GPU_ERROR(
      hipMemcpy(B, hB.data(), sizeof(htype) * ldb * K, hipMemcpyDefault));
  GPU_ERROR(
      hipMemcpy(C, hC.data(), sizeof(htype) * ldc * N, hipMemcpyDefault));

  size_t temp_storage_bytes = 0;
  d_temp_storage = NULL;

  if (self)
    Skyblas::dgemm<dtype, PARM, PARN>(temp_storage_bytes, d_temp_storage,
                                      blockCount, AOrder, BOrder, M, N, K,
                                      dalpha, A, lda, A, lda, dbeta, C, ldc);
  else
    Skyblas::dgemm<dtype, PARM, PARN>(temp_storage_bytes, d_temp_storage,
                                      blockCount, AOrder, BOrder, M, N, K,
                                      dalpha, A, lda, B, ldb, dbeta, C, ldc);

  GPU_ERROR(hipMalloc(&d_temp_storage, sizeof(htype) * temp_storage_bytes));

  if (self)
    Skyblas::dgemm<dtype, PARM, PARN>(temp_storage_bytes, d_temp_storage,
                                      blockCount, AOrder, BOrder, M, N, K,
                                      dalpha, A, lda, A, lda, dbeta, C, ldc);
  else
    Skyblas::dgemm<dtype, PARM, PARN>(temp_storage_bytes, d_temp_storage,
                                      blockCount, AOrder, BOrder, M, N, K,
                                      dalpha, A, lda, B, ldb, dbeta, C, ldc);

  GPU_ERROR(
      hipMemcpy(hC.data(), C, sizeof(htype) * ldc * N, hipMemcpyDefault));
  GPU_ERROR(
      hipMemcpy(C, hC2.data(), sizeof(htype) * ldc * N, hipMemcpyDefault));

  if (self)
    Skyblas::dgemm<dtype, PARM, PARN>(temp_storage_bytes, d_temp_storage,
                                      blockCount, AOrder, BOrder, M, N, K,
                                      dalpha, A, lda, A, lda, dbeta, C, ldc);
  else
    Skyblas::dgemm<dtype, PARM, PARN>(temp_storage_bytes, d_temp_storage,
                                      blockCount, AOrder, BOrder, M, N, K,
                                      dalpha, A, lda, B, ldb, dbeta, C, ldc);

  GPU_ERROR(
      hipMemcpy(hC2.data(), C, sizeof(htype) * ldc * N, hipMemcpyDefault));

  GPU_ERROR(hipDeviceSynchronize());

  if (self)
    cpuDgemm(AOrder, BOrder, M, N, K, halpha, hA.data(), lda, hA.data(), lda,
             hbeta, cpuC.data(), ldc);
  else
    cpuDgemm(AOrder, BOrder, M, N, K, halpha, hA.data(), lda, hB.data(), ldb,
             hbeta, cpuC.data(), ldc);

  bool passed = true;
  for (size_t n = 0; n < N; n++) {
    for (size_t m = 0; m < M; m++) {
      if (hC[n * ldc + m] != cpuC[n * ldc + m]) {
        cout << "\n( " << blockCount << " blocks, " << ((self) ? "A*A" : "A*B")
             << ") ";
        cout << "\e[31mMismatch\e[0m\n";

        printMatrix(hC, cpuC, N, M, ldc);
        cout << "--\n";
        printMatrix(hC2, hC, N, M, ldc, "\e[34m");
        cout << "--\n";
        printMatrix(cpuC, cpuC, N, M, ldc, "\e[0m");
        cout << "--\n\n";

        passed = false;
        break;
      }
    }
    if (!passed) break;
  }

  GPU_ERROR(hipFree(A));
  GPU_ERROR(hipFree(B));
  GPU_ERROR(hipFree(d_temp_storage));
  GPU_ERROR(hipFree(C));

  return passed;
}

int main(int argc, char **argv) {
  int sampleSize = 5;

  size_t M = PARM;
  size_t N = PARN;
  size_t K = (size_t)5 * 1024 * 1024 * 1024 / (M + N) / 8 * 0.02;

  cout << M << "xKx" << N << "\t";
  bool passed = true;
  for (size_t blockCount = 1 * 13; blockCount <= 8 * 13; blockCount += 2 * 13) {
    for (int t = 0; t < sampleSize; t++) {
      size_t lda = M + rand() % 4;
      size_t ldb = N + rand() % 4;
      size_t ldc = M + rand() % 4;
      if (M == N)
        passed &= testMatmul(Skyblas::COLUMN, Skyblas::ROW, M, N, K, lda, ldb,
                             ldc, blockCount, true);
      passed &= testMatmul(Skyblas::COLUMN, Skyblas::ROW, M, N, K, lda, ldb,
                           ldc, blockCount, false);
      cout << ".";
      cout.flush();
    }
  }
  if (passed) cout << "\e[32m Passed \e[0m\n";
  cout.flush();
}
