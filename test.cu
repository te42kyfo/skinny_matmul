#include <iostream>
#include <cstdlib>
#include <sys/time.h>
#include <iomanip>
#include <algorithm>
#include <vector>
#include <hip/hip_runtime.h>

#include "matmul.cuh"

using namespace std;

double dtime() {
  double tseconds = 0;
  struct timeval t;
  gettimeofday(&t, NULL);
  tseconds = (double)t.tv_sec + (double)t.tv_usec * 1.0e-6;
  return tseconds;
}

#define GPU_ERROR(ans) \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    cerr << "GPUassert: \"" << hipGetErrorString(code) << "\"  in " << file
         << ": " << line << "\n";
    if (abort) exit(code);
  }
}

void cpuMatmul(double *A, double *B, double *result, const size_t M,
               const size_t N, const size_t K) {
  for (size_t m = 0; m < M; m++) {
    for (size_t n = 0; n < N; n++) {
      double sum = 0;
      for (size_t k = 0; k < K; k++) {
        sum += A[k * M + m] * B[n * K + k];
      }
      result[n * M + m] = sum;
    }
  }
}

void testMatmul(const size_t M, const size_t N, const size_t K,
                const int blockCount) {
  double *A, *B, *d_temp_storage, *result;

  cout << "Setup, ";
  cout.flush();
  GPU_ERROR(hipMalloc(&A, sizeof(double) * M * K));
  GPU_ERROR(hipMalloc(&B, sizeof(double) * N * K));
  GPU_ERROR(hipMalloc(&result, sizeof(double) * M * N));

  vector<double> hA(M * K);
  vector<double> hB(N * K);
  vector<double> hResult(M * N, 0);
  vector<double> hResult2(M * N, 0);
  vector<double> cpuResult(M * N);

  static int salt = 0;
  srand(time(NULL) + salt++);

  for (size_t i = 0; i < M * K; i++) {
    hA[i] = rand() % 3 - 1;
  }
  for (size_t i = 0; i < N * K; i++) {
    hB[i] = rand() % 3 - 1;
  }

  GPU_ERROR(
      hipMemcpy(A, hA.data(), sizeof(double) * M * K, hipMemcpyDefault));
  GPU_ERROR(
      hipMemcpy(B, hB.data(), sizeof(double) * N * K, hipMemcpyDefault));

  size_t temp_storage_bytes = 0;
  matmul(temp_storage_bytes, NULL, A, B, NULL, M, N, K, blockCount);

  GPU_ERROR(hipMalloc(&d_temp_storage, sizeof(double) * temp_storage_bytes));

  cout << "GPU, ";
  cout.flush();

  matmul(temp_storage_bytes, d_temp_storage, A, B, result, M, N, K, blockCount);
  hipMemcpy(hResult.data(), result, sizeof(double) * M * N, hipMemcpyDefault);

  matmul(temp_storage_bytes, d_temp_storage, A, B, result, M, N, K, blockCount);
  hipMemcpy(hResult2.data(), result, sizeof(double) * M * N,
             hipMemcpyDefault);

  GPU_ERROR(hipDeviceSynchronize());

  cout << "CPU, ";
  cout.flush();
  cpuMatmul(hA.data(), hB.data(), cpuResult.data(), M, N, K);

  bool passed = true;
  for (size_t i = 0; i < N * M; i++) {
    if (hResult[i] != cpuResult[i]) {
      cout << "Mismatch\n";

      for (size_t n = 0; n < N; n++) {
        for (size_t m = 0; m < M; m++) {
          cout << hResult[n * M + m] << " \t";
        }
        cout << "\n";
      }
      cout << "--\n";

      for (size_t n = 0; n < N; n++) {
        for (size_t m = 0; m < M; m++) {
          cout << hResult2[n * M + m] << " \t";
        }
        cout << "\n";
      }
      cout << "--\n";

      for (size_t n = 0; n < N; n++) {
        for (size_t m = 0; m < M; m++) {
          cout << cpuResult[n * M + m] << " \t";
        }
        cout << "\n";
      }
      cout << "--\n\n";

      passed = false;
      break;
    }
  }
  if (passed) cout << "Passed (" << cpuResult[N * M / 2] << ")\n";

  GPU_ERROR(hipFree(A));
  GPU_ERROR(hipFree(B));
  GPU_ERROR(hipFree(d_temp_storage));
  GPU_ERROR(hipFree(result));
}

int main(int argc, char **argv) {
  int sampleSize = 5;

  for (size_t M = 1; M <= 10; M++) {
    size_t N = 1;
    size_t K = (size_t)5 * 1024 * 1024 * 1024 / (M + N) / 8 * 0.03;
    for (size_t blockCount = 13; blockCount < 8 * 13; blockCount += 13) {
      for (int t = 0; t < sampleSize; t++) {
        testMatmul(M, N, K, blockCount);
      }
      cout << M << "xKx" << N << "\t" << blockCount << endl;
    }
  }

  cout.flush();
}
