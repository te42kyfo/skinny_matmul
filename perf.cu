#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <algorithm>
#include <complex>
#include <cstdlib>
#include <functional>
#include <iomanip>
#include <iostream>
#include <vector>

#include "benchdb.hpp"
#include "cu_complex.h"
#include "dtime.hpp"
#include "gpu_error.cuh"
#include "types.hpp"
#include "versions.hpp"

#if !defined PARM || !defined PARN
#error "PARM or PARN is not specified! Specify M and N to measure"
#endif

using namespace std;

#ifdef TSMM
bool tsmttsm_mode = false;
bool tsmm_mode = true;
std::string multype = "TSMM";
#endif
#ifdef TSMTTSM
bool tsmttsm_mode = true;
bool tsmm_mode = false;
std::string multype = "TSMTTSM";
#endif

__global__ void initKernel(dtype* A, size_t N) {
  size_t tidx = blockDim.x * blockIdx.x + threadIdx.x;

  for (size_t idx = tidx; idx < N; idx += blockDim.x * gridDim.x) {
    A[idx] = MAKE_DTYPE(idx % 3 - 1, 0);
  }
}

dtype* A;
dtype* B;
dtype* C;
size_t totalA, totalB, totalC;

void initMatmul() {
  GPU_ERROR(hipMalloc(&A, sizeof(dtype) * totalA));
  GPU_ERROR(hipMalloc(&B, sizeof(dtype) * totalB));
  GPU_ERROR(hipMalloc(&C, sizeof(dtype) * totalC));
  initKernel<<<52, 256>>>(A, totalA);
  initKernel<<<52, 256>>>(B, totalB);
  initKernel<<<52, 256>>>(C, totalC);
}

void deInitMatmul() {
  GPU_ERROR(hipFree(A));
  GPU_ERROR(hipFree(B));
  GPU_ERROR(hipFree(C));
}

double measureMatmul(MatmulFunctionType matmulFunction, size_t M, size_t N,
                     size_t K, int lda, int ldb, int ldc, size_t blockCount,
                     int iters, bool self, dtype beta) {
  GPU_ERROR(hipDeviceSynchronize());

  bool passed = true;
  double t1 = dtime();
  for (int iter = 0; iter < iters; iter++) {
    if (!self) {
      passed = matmulFunction(blockCount, M, N, K, A, lda, makeDtype(2.0), B,
                              ldb, makeDtype(beta), C, ldc);
    } else if (tsmm_mode) {
      passed = matmulFunction(blockCount, M, N, K, C, ldc, makeDtype(2.0), B,
                              ldb, makeDtype(beta), C, ldc);
    } else if (M == N) {
      passed = matmulFunction(blockCount, M, N, K, A, lda, makeDtype(2.0), A,
                              lda, makeDtype(beta), C, ldc);
    } else {
      passed = false;
    }
  }
  GPU_ERROR(hipDeviceSynchronize());
  double t2 = dtime();
  double time = (t2 - t1) / iters;

  if (!passed)
    return -time;
  else
    return time;
}

int main(int argc, char** argv) {
  BenchDB db("benchmarks.db");

  hipDeviceProp_t prop;
  int deviceId;
  GPU_ERROR(hipGetDevice(&deviceId));
  GPU_ERROR(hipGetDeviceProperties(&prop, deviceId));
  std::string deviceName = prop.name;

  int m1 = 0;
  int m2 = 0;
  int n1 = 0;
  int n2 = 0;
  if (argc == 2) {
    m1 = 1;
    m2 = stoi(argv[1]);
  }
  if (argc >= 3) {
    m1 = stoi(argv[1]);
    m2 = stoi(argv[2]);
  }
  if (argc == 4) {
    cout << "Incomplete set of arguments\n";
    exit(1);
  }
  if (argc == 5) {
    n1 = stoi(argv[3]);
    n2 = stoi(argv[4]);
  }
  if (argc == 1) {
    m1 = m2 = PARM;
    n1 = n2 = PARN;
  }

  size_t maxMatrixSize = 1 * ((size_t)1 << 30) / (2 * sizeof(dtype));
  totalA = maxMatrixSize;

#ifdef TSMM
  auto versions = getEnabledTSMMVersions();
  MatmulFunctionType referenceFunction = tsmm_cublas<dtype>;
  totalB = 104 * 104;
  totalC = maxMatrixSize;
#endif
#ifdef TSMTTSM
  auto versions = getEnabledTSMTTSMVersions();
  MatmulFunctionType referenceFunction = tsmttsm_cublas<dtype>;
  totalB = maxMatrixSize;
  totalC = 104 * 104;
#endif
  initMatmul();

  for (int M = m1; M <= m2; M++) {
    for (int N = n1; N <= n2; N++) {
      if (n1 == 0 && n2 == 0) N = M;

      size_t lda = M;
#ifdef TSMM
      size_t ldb = M;
      size_t ldc = N;
      size_t maxK = maxMatrixSize / max(lda, ldb);
#endif
#ifdef TSMTTSM
      size_t ldb = N;
      size_t ldc = M;
      size_t maxK = maxMatrixSize / max(lda, ldb);
#endif

      for (const auto& matmulVersion : versions) {
        size_t K = 2000000;
        measureMatmul(matmulVersion.first, M, N, K, lda, ldb, ldc, 13, 1, false,
                      -1.0);
        double resultTime = measureMatmul(matmulVersion.first, M, N, K, lda,
                                          ldb, ldc, 13, 1, false, -1.0);

        while (resultTime > 0 && resultTime < 0.01 && K < maxK) {
          K = min(maxK, 2 * K);
          resultTime = measureMatmul(matmulVersion.first, M, N, K, lda, ldb,
                                     ldc, 13, 1, false, -1.0);
        }
        for (int self = 0; self <= (M == N || tsmm_mode ? 1 : 0); self++) {
          if (self == 1 && tsmm_mode) ldc = max(M, N);
          for (htype beta = (tsmm_mode ? 0.0 : 1.0); beta <= 1.0; beta += 1.0) {
            int iters = 1;

            double bestTime = -1;
            int bestBlockCount = 0;
            for (int blockCount = 1 * 13; blockCount <= 8 * 13;
                 blockCount += 13) {
              int sampleSize = 3;
              vector<double> times(sampleSize);
              for (int t = 0; t < sampleSize; t++) {
                times[t] =
                    measureMatmul(matmulVersion.first, M, N, K, lda, ldb, ldc,
                                  blockCount, iters, (self == 1), beta);
              }
              times.erase(remove_if(begin(times), end(times),
                                    [](double time) { return time < 0; }),
                          end(times));
              sort(times.begin(), times.end());

              if (times.size() != 0 &&
                  (times[sampleSize / 2] < bestTime || bestBlockCount == 0)) {
                bestTime = times[sampleSize / 2];
                bestBlockCount = blockCount;
              }
            }
            double flops = 0;
            double bw = 0;

            if (bestTime > 0) {
              if (tsmm_mode) {
                bw = ((beta == 0 || self == 1 ? 1.0 : 2.0) * N + M) * K *
                     sizeof(dtype) / bestTime * 1.0e-9;
                flops = (M + (beta == 0 ? 0 : 1)) * K * N * flopsPerCell /
                        bestTime * 1.0e-9;
              }
              if (tsmttsm_mode) {
                bw = (M + (self == 1 ? 0 : N)) * K * sizeof(dtype) / bestTime *
                     1.0e-9;
                flops = M * N * K * flopsPerCell / bestTime * 1.0e-9;
              }
            }
            cout << multype << " " << deviceName << " " << setw(3) << M << " "
                 << setw(3) << N << " " << setw(2) << beta << "    "
                 << (self == 1 ? "A*A" : "A*B") << "  " << matmulVersion.second
                 << " " << setw(9) << K << "  " << setw(8) << bestBlockCount
                 << " " << setprecision(3) << setw(8) << bestTime * 1000.0
                 << " " << setw(5) << setprecision(3) << flops << " " << setw(5)
                 << bw << "  \n";
            cout.flush();
            db.insert(multype, deviceName, types, M, N, matmulVersion.second,
                      self == 1, beta == 0, K, bestTime, flops, bw);
          }
        }
      }
      if (versions.size() > 1) cout << "\n";
    }
  }
  deInitMatmul();
}
