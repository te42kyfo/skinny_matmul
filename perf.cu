#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <sys/time.h>
#include <iomanip>
#include <algorithm>
#include <vector>
#include <hip/hip_runtime.h>

#include "matmul.cuh"

using namespace std;

double dtime() {
  double tseconds = 0;
  struct timeval t;
  gettimeofday(&t, NULL);
  tseconds = (double)t.tv_sec + (double)t.tv_usec * 1.0e-6;
  return tseconds;
}

#define GPU_ERROR(ans) \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    cerr << "GPUassert: \"" << hipGetErrorString(code) << "\"  in " << file
         << ": " << line << "\n";
    if (abort) exit(code);
  }
}

__global__ void initKernel(double *A, size_t N) {
  size_t tidx = blockDim.x * blockIdx.x + threadIdx.x;

  for (size_t idx = tidx; idx < N; idx += blockDim.x * gridDim.x) {
    A[idx] = 2.0;
  }
}

double measureMatmul(const size_t M, const size_t N, const size_t K,
                     const int blockCount) {
  double *A, *B, *d_temp_storage, *result;

  int iters = 3;
  GPU_ERROR(hipMalloc(&A, sizeof(double) * M * K));
  GPU_ERROR(hipMalloc(&B, sizeof(double) * N * K));
  initKernel << <52, 256>>> (A, M * K);
  initKernel << <52, 256>>> (B, N * K);

  size_t temp_storage_bytes = 0;
  matmul(temp_storage_bytes, NULL, A, B, NULL, M, N, K, blockCount);

  GPU_ERROR(hipMalloc(&d_temp_storage, sizeof(double) * temp_storage_bytes));
  GPU_ERROR(hipMalloc(&result, sizeof(double) * M * N));

  GPU_ERROR(hipDeviceSynchronize());
  double t1 = dtime();
  for (int iter = 0; iter < iters; iter++) {
    matmul(temp_storage_bytes, d_temp_storage, A, B, result, M, N, K,
           blockCount);
  }
  GPU_ERROR(hipDeviceSynchronize());
  double t2 = dtime();

  GPU_ERROR(hipFree(A));
  GPU_ERROR(hipFree(B));
  GPU_ERROR(hipFree(d_temp_storage));
  GPU_ERROR(hipFree(result));
  return (t2 - t1) / iters;
}

int main(int argc, char **argv) {
  int sampleSize = 5;

  size_t M = 2;
  size_t N = 2;
  size_t K = (size_t)5 * 1024 * 1024 * 1024 / (M + N) / 8 * 0.5;

  srand(time(NULL));

  for (size_t blockCount = 13; blockCount < 8 * 13; blockCount += 13) {
    vector<double> times(sampleSize);
    for (int t = 0; t < sampleSize; t++) {
      times[t] =
          measureMatmul(M, N, K + 2 * 1024 * (rand() % 1024), blockCount);
    }
    sort(times.begin(), times.end());

    cout << M << "xKx" << N << "\t" << setprecision(3) << blockCount << "\t"
         << (2 * M * N * K) * 1e-9 / times[sampleSize / 2] << std::endl
         << std::flush;
  }

  cout.flush();
}
