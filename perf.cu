#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <sys/time.h>
#include <iomanip>
#include <algorithm>
#include <vector>
#include <hip/hip_runtime.h>
#include <complex>

#include "skyblas.cuh"
#include "cu_complex.h"

#if !defined PARM || !defined PARN
#error "PARM or PARN is not specified! Specify M and N to measure"
#endif

using namespace std;


#define XSTR(s) STR(s)
#define STR(s) #s

#ifdef FC
typedef complex<float> htype;
typedef hipFloatComplex dtype;
dtype makeDtype(htype v) { return make_hipFloatComplex(v.real(), v.imag()); }
#define RAND_HTYPE(gen) htype(gen, gen)
#define MAKE_DTYPE(v1, v2) make_hipFloatComplex(v1, v2)

#elif DC
typedef complex<double> htype;
typedef hipDoubleComplex dtype;
dtype makeDtype(htype v) { return make_hipDoubleComplex(v.real(), v.imag()); }
#define RAND_HTYPE(gen) htype(gen, gen)
#define MAKE_DTYPE(v1, v2) make_hipDoubleComplex(v1, v2)

#elif FR
typedef float htype;
typedef float dtype;
dtype makeDtype(htype v) { return v; }
#define RAND_HTYPE(gen) htype(gen)
#define MAKE_DTYPE(v1, v2) float(v1)

#elif DR
typedef double htype;
typedef double dtype;
dtype makeDtype(htype v) { return v; }
#define RAND_HTYPE(gen) htype(gen)
#define MAKE_DTYPE(v1, v2) double(v1)

#endif

double dtime() {
  double tseconds = 0;
  struct timeval t;
  gettimeofday(&t, NULL);
  tseconds = (double)t.tv_sec + (double)t.tv_usec * 1.0e-6;
  return tseconds;
}

#define GPU_ERROR(ans) \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    cerr << "GPUassert: \"" << hipGetErrorString(code) << "\"  in " << file
         << ": " << line << "\n";
    if (abort) exit(code);
  }
}

__global__ void initKernel(dtype* A, size_t N) {
  size_t tidx = blockDim.x * blockIdx.x + threadIdx.x;

  for (size_t idx = tidx; idx < N; idx += blockDim.x * gridDim.x) {
    A[idx] = MAKE_DTYPE(idx % 3 - 1, 0);
  }
}

dtype* A;
dtype* B;
dtype* C;
dtype* d_temp_storage;

size_t temp_storage_bytes;

void initMatmul(Skyblas::MEMORY_ORDER AOrder, Skyblas::MEMORY_ORDER BOrder,
                int M, int N, int K, int lda, int ldb, int ldc,
                size_t blockCount) {
  GPU_ERROR(hipMalloc(&A, sizeof(dtype) * lda * K));
  GPU_ERROR(hipMalloc(&B, sizeof(dtype) * ldb * K));
  GPU_ERROR(hipMalloc(&C, sizeof(dtype) * ldc * N));
  initKernel<<<52, 256>>>(A, lda * K);
  initKernel<<<52, 256>>>(B, ldb * K);
  initKernel<<<52, 256>>>(C, ldc * N);

  temp_storage_bytes = 0;
  d_temp_storage = NULL;

  Skyblas::dgemm<dtype, PARM, PARN>(
      temp_storage_bytes, d_temp_storage, blockCount, AOrder, BOrder, M, N, K,
      makeDtype(1.0), A, lda, B, ldb, makeDtype(1.0), C, ldc);

  GPU_ERROR(hipMalloc(&d_temp_storage, sizeof(dtype) * temp_storage_bytes));
}

void deInitMatmul() {
  GPU_ERROR(hipFree(A));
  GPU_ERROR(hipFree(B));
  GPU_ERROR(hipFree(C));
  GPU_ERROR(hipFree(d_temp_storage));
}

double measureMatmul(Skyblas::MEMORY_ORDER AOrder, Skyblas::MEMORY_ORDER BOrder,
                     size_t M, size_t N, size_t K, int lda, int ldb, int ldc,
                     size_t blockCount, int iters, bool self) {
  GPU_ERROR(hipDeviceSynchronize());

  htype halpha = 1.0;
  htype hbeta = 2.0;

  dtype dalpha = makeDtype(halpha);
  dtype dbeta = makeDtype(hbeta);

  double t1 = dtime();
  for (int iter = 0; iter < iters; iter++) {
    if (self)
      Skyblas::dgemm<dtype, PARM, PARN>(temp_storage_bytes, d_temp_storage,
                                        blockCount, AOrder, BOrder, M, N, K,
                                        dalpha, A, lda, A, lda, dbeta, C, ldc);
    else
      Skyblas::dgemm<dtype, PARM, PARN>(temp_storage_bytes, d_temp_storage,
                                        blockCount, AOrder, BOrder, M, N, K,
                                        dalpha, A, lda, B, ldb, dbeta, C, ldc);
  }
  GPU_ERROR(hipDeviceSynchronize());
  double t2 = dtime();

  return (t2 - t1) / iters;
}

int main(int argc, char** argv) {
  size_t N = PARN;
  size_t M = PARM;
  bool self = false;

  if (M == 0 || N == 0) {
    std::cout << "  M   N         K  self  blockcount     time  perf\n";
    return 0;
  }

  size_t maxK = 2 * ((size_t)1 << 30) / ((M + N) * 8);
  size_t K = 0.2 * ((size_t)1 << 30) / ((M + N) * 8);

  initMatmul(Skyblas::COLUMN, Skyblas::ROW, M, N, maxK, M, N, N, 8 * 13);

  double resultTime = 0;
  while (resultTime < 0.1 && K * 2 <= maxK) {
    K *= 2;
    resultTime = measureMatmul(Skyblas::COLUMN, Skyblas::ROW, M, N, K, M, N, M,
                               26, 1, false);
  }

  int iters = max(1, (int)(0.05 / resultTime));

  size_t lda = M;
  double bestTime = 0;
  int bestBlockCount = 0;
  for (int blockCount = 1 * 13; blockCount <= 8 * 13; blockCount += 13) {
    int sampleSize = 3;
    vector<double> times(sampleSize);
    for (int t = 0; t < sampleSize; t++) {
      times[t] = measureMatmul(Skyblas::COLUMN, Skyblas::ROW, M, N, K, lda, N,
                               M, blockCount, iters, self);
    }
    sort(times.begin(), times.end());

    if (times[sampleSize / 2] < bestTime || bestBlockCount == 0) {
      bestTime = times[sampleSize / 2];
      bestBlockCount = blockCount;
    }
  }

  cout << XSTR(PREC) << " " << XSTR(MODE) << " " << setw(3) << M << " "
       << setw(3) << N << " " << setw(9) << K << "  "
       << ((self) ? "true " : "false") << " " << setw(10) << bestBlockCount
       << " " << setprecision(3) << setw(8) << bestTime << " " << setw(5)
       << M * N * K * 2 / bestTime * 1e-9 << "\n";
  cout.flush();

  deInitMatmul();
}
